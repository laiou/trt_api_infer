#include "hip/hip_runtime.h"
#include "yololayer.h"
#include "utils.h"
#include <assert.h>

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin()
    {
        mClassCount = CLASS_NUM;
        mYoloKernel.clear();
        mYoloKernel.push_back(yolo1);
        mYoloKernel.push_back(yolo2);
        mYoloKernel.push_back(yolo3);

        mKernelCount = mYoloKernel.size();
    }
    
    YoloLayerPlugin::~YoloLayerPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(mYoloKernel.data(),d,kernelSize);
        d += kernelSize;

        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer) const
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(d,mYoloKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t YoloLayerPlugin::getSerializationSize() const
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount)  + sizeof(Yolo::YoloKernel) * mYoloKernel.size();
    }

    int YoloLayerPlugin::initialize()
    { 
        return 0;
    }
    
    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalsize = MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void YoloLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void YoloLayerPlugin::detachFromContext() {}

    const char* YoloLayerPlugin::getPluginType() const
    {
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void YoloLayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloLayerPlugin::clone() const
    {
        YoloLayerPlugin *p = new YoloLayerPlugin();
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1.0f / (1.0f + expf(-data)); };
    //执行相应的操作
    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
        //计算线程id，这里每一个线程处理一个grid上的预测（整个batch的数据一共是有yolo.width*yolo.height*batchSize个grid的）
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;
        //统计当前yolo层图像划分的grid个数
        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        //获取当前的线程偏移量，也就是当前在某张图片的哪一个grid上
        idx = idx - total_grid*bnIdx;
        int info_len_i = 5 + classes;
        //定位当前输入的位置，也就是一个batch中的第几张图数据的起始位置
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);
        //遍历不同的anchor上的数据
        for (int k = 0; k < 3; ++k) {
            int class_id = 0;
            float max_cls_prob = 0.0;
            //然后遍历每一个预测值
            for (int i = 5; i < info_len_i; ++i) {
            //通过logist进行计算类别置信度
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                //更新max_cla_prob
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    //之所以要减去5的原因前面一开始i是从5开始的，这里需要计算对应的class_id
                    class_id = i - 5;
                }
            }
            //然后在对相应的box输出进行logist转换，得到box的阈值
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            //如果类别置信度或者box置信度小于阈值，就可以放弃了
            if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH) continue;
            //定位对应的保存输出的位置
            float *res_count = output + bnIdx*outputElem;
            //然后将相应的输出值抽取到output中
            int count = (int)atomicAdd(res_count, 1);
            if (count >= MAX_OUTPUT_BBOX_COUNT) return;
            char* data = (char * )res_count + sizeof(float) + count*sizeof(Detection);
            Detection* det =  (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            det->bbox[0] = (col + Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
            det->bbox[1] = (row + Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
            det->bbox[2] = expf(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
            det->bbox[3] = expf(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
            det->det_confidence = box_prob;
            det->class_id = class_id;
            det->class_confidence = max_cls_prob;
        }
    }

    void YoloLayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {

        void* devAnchor;
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        //分配相应的内存
        CUDA_CHECK(hipMalloc(&devAnchor,AnchorLen));
        //输出元素的数量
        int outputElem = 1 + MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        //循环给每一个batch的数据的输出分配内存
        for(int idx = 0 ; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx*outputElem, 0, sizeof(float)));
        }
        int numElem = 0;
        //循环处理每一个yolo层
        for (unsigned int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            //计算不同层输入的数据量，实际上也就是一个batchsize上全部的grid数量
            numElem = yolo.width*yolo.height*batchSize;
            //判断输入数据是否超出默认的线程数量
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            //将相应的数据从主机内存复制到设备内存，也就是将当前的yolo层的anchor数据从yolo.anchors传输到devAnchor
            CUDA_CHECK(hipMemcpy(devAnchor, yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
            //进行具体操作的核函数，整个yolo层完成的就是提取每一个grid下面不同尺度的anchor上的预测框，判断是否超过阈值，然后将相应满足
            //条件的框计算对应的框的置信度和类别置信度，然后将结果存入output中
            CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                (inputs[i],output, numElem, yolo.width, yolo.height, (float *)devAnchor, mClassCount ,outputElem);
        }

        CUDA_CHECK(hipFree(devAnchor));
    }

    //这里看一下具体的实现yolo层的操作
    int YoloLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        //具体的yolo层的gpu前向传播
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);

        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const
    {
            return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        YoloLayerPlugin* obj = new YoloLayerPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}
